// Includes
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <ctime>

// Definitions
#define M_PI 3.14276
#define c 299792458
#define mu0 M_PI*4e-7
#define eta0 c*mu0

void checkError(hipError_t cudaStatus)
{
    // [--------------- GPU error checking ---------------]
    if (cudaStatus != hipSuccess) { // throws any errors encountered
        std::cout << stderr << " :error code: " << cudaStatus << std::endl;
        exit(1);
    }
}

__global__ void zeroesKernel(double* v1, double* v2, double* v3, double* v4, const int n) {
    // divide work amongst threads and blocks
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = blockDim.x * gridDim.x;
    //*/
    for (size_t i = tid + stride; i < n; i += stride) {
        v1[i] = 0;
        v2[i] = 0;
        v3[i] = 0;
        v4[i] = 0;
        __syncthreads();
    }
    //*/
}


void stageSource(double* V1, double* V2, double* V3, double* V4, int x, int y, double E0, int NY) {
    /* Stage 1: Source */

    // Adapted to be 1D
    V1[x * NY + y] = V1[x * NY + y] + E0;
    V2[x * NY + y] = V2[x * NY + y] - E0;
    V3[x * NY + y] = V3[x * NY + y] - E0;
    V4[x * NY + y] = V4[x * NY + y] + E0;
    // Using 1 dimensional arrays is more obvious to work with when porting to GPU

}

__global__ void sourceKernel(double* V1, double* V2, double* V3, double* V4, const int x, const int y, const double E0, const int NY) {
    /* Stage 1: Source */

    V1[x * NY + y] = V1[x * NY + y] + E0;
    V2[x * NY + y] = V2[x * NY + y] - E0;
    V3[x * NY + y] = V3[x * NY + y] - E0;
    V4[x * NY + y] = V4[x * NY + y] + E0;

    
} // end func

void stageScatter(double* V1, double* V2, double* V3, double* V4, int NX, int NY, double Z) {
    /* Stage 2: Scatter */
    // Variables 
    double I = 0, V = 0;

    // Parallelisable code

    // for int i = 0; i < NX*NY; i++
    for (int x = 0; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            I = (2 * V1[(x * NY) + y] + 2 * V4[(x * NY) + y] - 2 * V2[(x * NY) + y] - 2 * V3[(x * NY) + y]) / (4 * Z);

            V = 2 * V1[x * NY + y] - I * Z;         //port1
            V1[x * NY + y] = V - V1[x * NY + y];

            V = 2 * V2[x * NY + y] + I * Z;         //port2
            V2[x * NY + y] = V - V2[x * NY + y];

            V = 2 * V3[x * NY + y] + I * Z;         //port3
            V3[x * NY + y] = V - V3[x * NY + y];

            V = 2 * V4[x * NY + y] - I * Z;         //port4
            V4[x * NY + y] = V - V4[x * NY + y];
        }
    }
}

__global__ void scatterKernel(double* V1, double* V2, double* V3, double* V4, const int NX, const int NY, const double Z) {
    // Variables
    double I = 0, V = 0;
    // Thread identities
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = blockDim.x * gridDim.x;
    //*/
    for (size_t i = tid + stride; i < NX*NY; i += stride) {
        V = 2 * V1[i] - I * Z;         //port1
        V1[i] = V - V1[i];

        V = 2 * V2[i] + I * Z;         //port2
        V2[i] = V - V2[i];

        V = 2 * V3[i] + I * Z;         //port3
        V3[i] = V - V3[i];

        V = 2 * V4[i] - I * Z;         //port4
        V4[i] = V - V4[i];
    
    }
}

void stageConnect(double* V1, double* V2, double* V3, double* V4, int NX, int NY, double rXmin, double rXmax, double rYmin, double rYmax) {
    /* Stage 3: Connect */
    // Variables
    double tempV = 0;

    // Connect internals
    for (int x = 1; x < NX; x++) {
        for (int y = 0; y < NY; y++) {
            tempV = V2[x * NY + y];
            V2[x * NY + y] = V4[(x - 1) * NY + y];
            V4[(x - 1) * NY + y] = tempV;
        }
    }
    for (int x = 0; x < NX; x++) {
        for (int y = 1; y < NY; y++) {
            tempV = V1[x * NY + y];
            V1[x * NY + y] = V3[x * NY + y - 1];
            V3[x * NY + y - 1] = tempV;
        }
    }

    // Connect boundaries
    for (int x = 0; x < NX; x++) {
        V3[x * NY + NY - 1] = rYmax * V3[x * NY + NY - 1];
        V1[x * NY] = rYmin * V1[x * NY]; // V1[x * NY + 0] = rYmin * V1[x * NY + 0];
    }
    for (int y = 0; y < NY; y++) {
        V4[(NX - 1) * NY + y] = rXmax * V4[(NX - 1) * NY + y];
        V2[y] = rXmin * V2[y]; // V2[0 * NY + y] = rXmin * V2[0 * NY + y];
    }
}


int main() {

    // Start timer
    std::clock_t start = std::clock();

    /* Variables */
    // Changable variables
    int NX = 100; // number of X
    int NY = 100; // number of Y
    int NT = 8192; // number of Times/Iterations
    double dl = 1;

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0); // GPU interrogation

    double dt = dl / (sqrt(2.) * c);
    double* v1;
    double* v2;
    double* v3;
    double* v4; // send to GPU
    
    // Retrieval from GPU
    double* V1 = new double[int(NX * NY)];
    double* V2 = new double[int(NX * NY)];
    double* V3 = new double[int(NX * NY)];
    double* V4 = new double[int(NX * NY)];
    
    // Scatter Coefficient
    double Z = eta0 / sqrt(2.);

    // Boundary connect Coefficiants
    double rXmin = -1;
    double rXmax = -1;
    double rYmin = -1;
    double rYmax = -1;

    // input parameters
    double width = 20 * dt * sqrt(2.);
    double delay = 100 * dt * sqrt(2.);
    int Ein[] = { 10,10 };
    // output parameters
    int Eout[] = { 15,15 };

    // file output
    std::ofstream output("output.out");


    // Initialise GPU
    cudaStatus = hipDeviceSynchronize();
    checkError(cudaStatus);

    cudaStatus = hipMalloc(&v1, NX * NY * sizeof(double)); // Memory allocate for points array
    checkError(cudaStatus);
    cudaStatus = hipMalloc(&v2, NX * NY * sizeof(double)); // Memory allocate for points array
    checkError(cudaStatus);
    cudaStatus = hipMalloc(&v3, NX * NY * sizeof(double)); // Memory allocate for points array
    checkError(cudaStatus);
    cudaStatus = hipMalloc(&v4, NX * NY * sizeof(double)); // Memory allocate for points array
    checkError(cudaStatus);

    cudaStatus = hipDeviceSynchronize();
    checkError(cudaStatus);

    // Zero values on GPU - faster than copying array data


    for (int n = 0; n < NT; n++) {
        // Variables dependant on n
        double E0 = (1 / sqrt(2.)) * exp(-(n * dt - delay) * (n * dt - delay) / (width * width));

        /* Stage 1: Source */
        stageSource(V1, V2, V3, V4, Ein[0], Ein[1], E0, NY);

        /* Stage 2: Scatter */
        stageScatter(V1, V2, V3, V4, NX, NY, Z);

        /* Stage 3: Connect */
        stageConnect(V1, V2, V3, V4, NX, NY, rXmin, rXmax, rYmin, rYmax);

        output << n * dt << "  " << V2[Eout[0] * NY + Eout[1]] + V4[Eout[0] * NY + Eout[1]] << std::endl;
        if (n % 100 == 0)
            std::cout << n << std::endl;

    }
    output.close();
    std::cout << "Done: " << ((std::clock() - start) / (double)CLOCKS_PER_SEC) << std::endl;
    std::cin.get();


}

